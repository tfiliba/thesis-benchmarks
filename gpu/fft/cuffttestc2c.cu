#include <stdlib.h>
#include <stdio.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>
//#include <cutil_inline.h>
#include <helper_timer.h>


//#define NX      256
//#define BATCH   4
#define MIN_NX      256
#define MAX_NX      16777216
#define MIN_BATCH   1
#define MAX_BATCH   4096
#define MAX_DIM     16777216*4
//#define MAX_DIM     16777216

#define AVERAGED_ITERATIONS 100


static hipfftHandle plan;
hipfftComplex *gpudata;
hipfftComplex *fftgpudata;


int main ()
{
    //int deviceCount = 0;
    hipError_t err = hipInit(0);
    //CU_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));
    //printf("There are %d devices supporting CUDA\n", deviceCount);
    
    long long i;
    long long nx;
    long long batch;
    StopWatchInterface * complete_fft_timer;
    StopWatchInterface * complete_fft_timer2;
    StopWatchInterface * piecewise_fft_timer;
    StopWatchInterface * copy_to_gpu_timer;
    StopWatchInterface * fft_only_timer;
    StopWatchInterface * copy_from_gpu_timer;
    
    sdkCreateTimer(&complete_fft_timer);
    sdkCreateTimer(&complete_fft_timer2);
    sdkCreateTimer(&piecewise_fft_timer);
    sdkCreateTimer(&copy_to_gpu_timer);
    sdkCreateTimer(&fft_only_timer);
    sdkCreateTimer(&copy_from_gpu_timer);
    
    //hipfftHandle plan;
    hipfftComplex *data;
    hipfftComplex *result;
    //hipfftComplex *gpudata;
    
    hipHostMalloc(&data, sizeof(hipfftComplex)*MAX_DIM);
    hipHostMalloc(&result, sizeof(hipfftComplex)*MAX_DIM);
    //hipHostAlloc(&data, sizeof(hipfftComplex)*MAX_DIM,hipHostMallocWriteCombined);
    //hipHostAlloc(&result, sizeof(hipfftComplex)*MAX_DIM,hipHostMallocWriteCombined);
    
    // get the total global memory of the device
    hipDevice_t dev=0;
    size_t totalGlobalMem;
	hipDeviceTotalMem(&totalGlobalMem, dev);
    fprintf(stderr,"Total amount of global memory: %u bytes\n", totalGlobalMem);
    
    //fprintf(stderr, "Initializing data... ");
    // generate some random data
    for(i=0; i<MAX_DIM; i++)
    {
        data[i].x=1.0f;
        data[i].y=1.0f;
    }
    //fprintf(stderr, "done\n");

    fprintf(stderr, "Testing cufft C2C 1D fft with hipHostMalloc allocated memory\n");
    //fprintf(stderr, "nx\tbatch\ttime\tcopy_to_gpu\tactual_fft\tcopy_from_gpu\tavg\n");
    for(nx=MIN_NX; nx<=MAX_NX; nx=nx*2)
    //for(nx=4096; nx<=MAX_NX; nx+=4096)
    {
        fprintf(stderr, "Pts\tbatch\ttotal time\ttotal time 100x\tsum piecewise\tcopy to gpu\tfft on gpu\tcopy from gpu\ttimes reported in ms\n");
        for(batch=MIN_BATCH;batch<=MAX_BATCH;batch=batch*2)
        {
            if(/*sizeof(hipfftComplex)*nx*batch*2 <= totalGlobalMem/2 &&*/ nx*batch<MAX_DIM)
            {  
                //fprintf(stderr, "Allocating %lld bytes\n", sizeof(hipfftComplex)*nx*batch*2);
                // allocate device memory for the fft
                hipMalloc((void**)&gpudata,sizeof(hipfftComplex)*nx*batch);
                hipMalloc((void**)&fftgpudata,sizeof(hipfftComplex)*nx*batch);

                hipfftPlan1d(&plan,nx,HIPFFT_C2C, batch);
                hipDeviceSynchronize();
                
                sdkResetTimer(&complete_fft_timer);
                sdkStartTimer(&complete_fft_timer);
                // run the fft
                // allocate device memory and copy over data
                hipMemcpy(gpudata, data, sizeof(hipfftComplex)*nx*batch, hipMemcpyHostToDevice);
                hipDeviceSynchronize();
                // run the fft
                hipfftExecC2C(plan,gpudata,fftgpudata,HIPFFT_FORWARD);
                hipDeviceSynchronize();
                // copy the result back
                hipMemcpy(result, fftgpudata, sizeof(hipfftComplex)*nx*batch, hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                sdkStopTimer(&complete_fft_timer);
                
                
                
                sdkResetTimer(&complete_fft_timer2);
                sdkStartTimer(&complete_fft_timer2);
                for(int fftiter=0;fftiter<AVERAGED_ITERATIONS;fftiter++)
                {
                    // run the fft
                    // allocate device memory and copy over data
                    //hipMemcpy(gpudata, data, sizeof(hipfftComplex)*nx*batch, hipMemcpyHostToDevice);
                    hipDeviceSynchronize();
                    // run the fft
                    hipfftExecC2C(plan,gpudata,fftgpudata,HIPFFT_FORWARD);
                    //hipDeviceSynchronize();
                    // copy the result back
                    hipMemcpy(result, fftgpudata, sizeof(hipfftComplex)*nx*batch, hipMemcpyDeviceToHost);
                }
                hipDeviceSynchronize();
                sdkStopTimer(&complete_fft_timer2);
                
                
                
                sdkResetTimer(&piecewise_fft_timer);
                sdkStartTimer(&piecewise_fft_timer);
                
                sdkResetTimer(&copy_to_gpu_timer);
                sdkStartTimer(&copy_to_gpu_timer);
                hipMemcpy(gpudata, data, sizeof(hipfftComplex)*nx*batch, hipMemcpyHostToDevice);
                hipDeviceSynchronize();
                sdkStopTimer(&copy_to_gpu_timer);
                
                sdkResetTimer(&fft_only_timer);
                sdkStartTimer(&fft_only_timer);
                hipfftExecC2C(plan,gpudata,fftgpudata,HIPFFT_FORWARD);
                hipDeviceSynchronize();
                sdkStopTimer(&fft_only_timer);
                
                sdkResetTimer(&copy_from_gpu_timer);
                sdkStartTimer(&copy_from_gpu_timer);
                hipMemcpy(result, fftgpudata, sizeof(hipfftComplex)*nx*batch, hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                sdkStopTimer(&copy_from_gpu_timer);
                
                sdkStopTimer(&piecewise_fft_timer);
                
                hipfftDestroy(plan);
                hipFree(gpudata);
                hipFree(fftgpudata);
                hipDeviceSynchronize();
            
                
                fprintf(stderr, "%lld\t%lld\t%f\t%f\t%f\t%f\t%f\t%f\t%f\t%d\n",
                    nx, batch, 
                    sdkGetTimerValue(&complete_fft_timer), sdkGetTimerValue(&complete_fft_timer2)/AVERAGED_ITERATIONS, sdkGetTimerValue(&piecewise_fft_timer), 
                    sdkGetTimerValue(&copy_to_gpu_timer), 
                    sdkGetTimerValue(&fft_only_timer), sdkGetTimerValue(&copy_from_gpu_timer),
                    /*sdkGetTimerValue(&complete_fft_timer)/(nx*batch),*/ sizeof(hipfftComplex)*nx*batch);
            }
        }
    }
    
    //print fft data
//    for(i=0; i<NX*BATCH; i++)
//    {
//        fprintf(stderr,"%d %f %f\n", i, data[i].x, data[i].y);
//    }
    
    
    return 0;
}
